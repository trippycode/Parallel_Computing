/*
* This is the cuda C program for the dot product of two matrices.
We will compile this cuda script using the NVCC compiler and save it as .cu file

* CUDA-C program and library source code filenames always use the ​ .cu ​ file
extension.


*/

// File that has declaration types of all the particular CUDA datatypes.
#include<hip/hip_runtime.h>

// For standard IO functions
#include<stdio.h>

// For using the malloc and free dynamic memory allocation
#include<stdlib.h>

// create our own version of the np.allclose
// EPSILON is the delta value to consider two values similar
#define _EPSILON 0.001
#define _ABS(x)(x>0.0f ? x:-x)

/*
We can now create our own version of ​ allclose​ . This will take in two float
pointers and an integer value, ​ len​ . We loop through both arrays and check
them: if any points differ by more than _ ​ EPSILON​ , we return -1, otherwise we
return 0 to indicate that the two arrays do indeed match.
*/

__host__ int allclose(float *A, float *B, int len)
{

	int returnval = 0;

	for (int i = 0; i < len; i++)
	{
		if ( _ABS(A[i] - B[i]) > _EPSILON )
		{
			returnval = -1;
			break;
		}
	}

	return(returnval);
}


// row-column dot-product for matrix multiplication
__device__ float rowcol_dot(float *matrix_a, float *matrix_b, int row, int col, int N)
{
	float val = 0;

	for (int k=0; k < N; k++)
	{
        val += matrix_a[ row*N + k ] * matrix_b[ col + k*N];
	}

	return(val);
}

// matrix multiplication kernel that is parallelized over row/column tuples.
__global__ void matrix_mult_ker(float * matrix_a, float * matrix_b, float * output_matrix, int N)
{

    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

	output_matrix[col + row*N] = rowcol_dot(matrix_a, matrix_b, row, col, N);
}


__host__ int main()
{

	// Initialize to use first GPU.
	hipSetDevice(0);

	// this indicates the width/height of the matrices
	int N = 4;

	// this will indicate how many bytes to allocate to store a test or output matrix
	int num_bytes = sizeof(float)*N*N;

	// input test matrix A
	float h_A[] = {	1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0 };

	// input test matrix B
	float h_B[] = {	14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0 };

	// expected output of A times B
	float h_AxB[] = { 140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0 };


	// these pointers will be used for the GPU.
	// (notice how we use normal float pointers)
	float * d_A;
	float * d_B;
	float * d_output;

	// allocate memory for the test matrices on the GPU
	hipMalloc((float **) &d_A, num_bytes);
	hipMalloc((float **) &d_B, num_bytes);

	// copy the test matrices to the GPU
	hipMemcpy(d_A, h_A, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, num_bytes, hipMemcpyHostToDevice);

	// allocate memory for output on GPU
	hipMalloc((float **) &d_output, num_bytes);

	// this will store the output from the GPU
	float * h_output;
	h_output = (float *) malloc(num_bytes);

	// setup our block and grid launch parameters with the dim3 class.
	dim3 block(2,2,1);
	dim3 grid(2,2,1);

	// launch our kernel
	matrix_mult_ker <<< grid, block >>> (d_A, d_B, d_output, N);

	// synchronize on the host, to ensure our kernel has finished executing.
	hipDeviceSynchronize();

	// copy output from device to host.
	hipMemcpy(h_output, d_output, num_bytes, hipMemcpyDeviceToHost);

	// synchronize again.
	hipDeviceSynchronize();

	// free arrays on device.
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_output);

	// reset the GPU.
	hipDeviceReset();


	// Check to see if we got the expected output.
	// in both cases, remember to de-allocate h_output before returning.

	if (allclose(h_AxB, h_output, N*N) < 0)
	{
		printf("Error!  Output of kernel does not match expected output.\n");
		free(h_output);
		return(-1);
	}
	else
	{
		printf("Success!  Output of kernel matches expected output.\n");
		free(h_output);
		return(0);
	}


}